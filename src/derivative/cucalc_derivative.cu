#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <stdio.h>

#include <iostream>

#include "cucalc/cucalc.h"
#include "cucalc/cucalc_common.h"
#include "cucalc/cucalc_integration.h"

__global__ void cucalc_function_calculate(void *func, double h, double *d_fx, double a, size_t thread_count){
  size_t tid = threadIdx.x + blockIdx.x * blockDim.x;
  double x = tid*h + a; 
  double result = (cucalc_func(func))(x);
  d_fx[tid] = result;
}

__global__ void cucalc_derivative_backward(double h, double *d_fx,double *d_fx_out) {

  size_t tid = threadIdx.x + blockIdx.x * blockDim.x;
  if(tid != 0){
    d_fx_out[tid-1] = (d_fx[tid] - d_fx[tid - 1])/h;
  } 
}
__global__ void cucalc_derivative_forward(double h, double *d_fx,double *d_fx_out) {
  
  size_t tid = threadIdx.x + blockIdx.x * blockDim.x;
  if(tid != blockDim.x){
    d_fx_out[tid] = (d_fx[tid + 1] - d_fx[tid])/h;

}
}
__global__ void cucalc_derivative_central(double h, double *d_fx,double *d_fx_out) {
  
  size_t tid = threadIdx.x + blockIdx.x * blockDim.x;
  if(tid != 0 && tid != blockDim.x){
    d_fx_out[tid-1] = (d_fx[tid + 1] - d_fx[tid-1])/h;
}
}

double* cucalc_derivative_backward(void *func, double a, double b, size_t steps) {
  hipError_t cuda_ret;
  int BLOCK_SIZE = 512;
  size_t thread_count = steps + 2;
  dim3 blockSize(BLOCK_SIZE, 1, 1);
  dim3 gridSize((thread_count - 1) / BLOCK_SIZE + 1, 1, 1);
  double h = (b - a) / steps;
  a = a - h; //for 1 backward

  double *d_fx, *d_fx_out,*h_fx;
  cuda_ret = hipMalloc((void **)&d_fx, sizeof(double) * thread_count);
  cudaErrorCheck(cuda_ret, "Unable to allocate memory on GPU", 1);

  cuda_ret = hipMalloc((void **)&d_fx_out, sizeof(double) * thread_count);
  cudaErrorCheck(cuda_ret, "Unable to allocate memory on GPU", 1);

  cuda_ret = hipHostMalloc((void **)&h_fx, sizeof(double) * thread_count);
  cudaErrorCheck(cuda_ret, "Unable to allocate memory on host", 1);

  cucalc_function_calculate<<<gridSize, blockSize>>>(func, h, d_fx, a, thread_count+1);
  cucalc_derivative_backward<<<gridSize, blockSize>>>(h, d_fx, d_fx_out);

  cuda_ret = hipMemcpy(h_fx, d_fx_out, thread_count * sizeof(double), hipMemcpyDeviceToHost);
  cudaErrorCheck(cuda_ret, "Unable to memory to host", 1);

  cuda_ret = hipDeviceSynchronize();
  cudaErrorCheck(cuda_ret, "Unable to launch kernel", 1);

  return h_fx;
}

double* cucalc_derivative_forward(void *func, double a, double b, size_t steps) {
  hipError_t cuda_ret;
  int BLOCK_SIZE = 512;
  size_t thread_count = steps + 2;
  dim3 blockSize(BLOCK_SIZE, 1, 1);
  dim3 gridSize((thread_count - 1) / BLOCK_SIZE + 1, 1, 1);
  double h = (b - a) / steps;

  double *d_fx, *d_fx_out,*h_fx;
  cuda_ret = hipMalloc((void **)&d_fx, sizeof(double) * thread_count);
  cudaErrorCheck(cuda_ret, "Unable to allocate memory on GPU", 1);

  cuda_ret = hipMalloc((void **)&d_fx_out, sizeof(double) * thread_count);
  cudaErrorCheck(cuda_ret, "Unable to allocate memory on GPU", 1);

  cuda_ret = hipHostMalloc((void **)&h_fx, sizeof(double) * thread_count);
  cudaErrorCheck(cuda_ret, "Unable to allocate memory on host", 1);

  cucalc_function_calculate<<<gridSize, blockSize>>>(func, h, d_fx, a, thread_count+1);
  cucalc_derivative_backward<<<gridSize, blockSize>>>(h, d_fx, d_fx_out);

  cuda_ret = hipMemcpy(h_fx, d_fx_out, thread_count * sizeof(double), hipMemcpyDeviceToHost);
  cudaErrorCheck(cuda_ret, "Unable to memory to host", 1);

  cuda_ret = hipDeviceSynchronize();
  cudaErrorCheck(cuda_ret, "Unable to launch kernel", 1);

  return h_fx;
}

double* cucalc_derivative_central(void *func, double a, double b, size_t steps) {
  hipError_t cuda_ret;
  int BLOCK_SIZE = 512;
  size_t thread_count = steps + 2;
  dim3 blockSize(BLOCK_SIZE, 1, 1);
  dim3 gridSize((thread_count - 1) / BLOCK_SIZE + 1, 1, 1);
  double h = (b - a) / steps;
  a = a - h; //for 1 backward

  double *d_fx, *d_fx_out,*h_fx;
  cuda_ret = hipMalloc((void **)&d_fx, sizeof(double) * thread_count);
  cudaErrorCheck(cuda_ret, "Unable to allocate memory on GPU", 1);

  cuda_ret = hipMalloc((void **)&d_fx_out, sizeof(double) * thread_count);
  cudaErrorCheck(cuda_ret, "Unable to allocate memory on GPU", 1);

  cuda_ret = hipHostMalloc((void **)&h_fx, sizeof(double) * thread_count);
  cudaErrorCheck(cuda_ret, "Unable to allocate memory on host", 1);

  cucalc_function_calculate<<<gridSize, blockSize>>>(func, h, d_fx, a, thread_count+2);
  cucalc_derivative_backward<<<gridSize, blockSize>>>(h, d_fx, d_fx_out);

  cuda_ret = hipMemcpy(h_fx, d_fx_out, thread_count * sizeof(double), hipMemcpyDeviceToHost);
  cudaErrorCheck(cuda_ret, "Unable to memory to host", 1);

  cuda_ret = hipDeviceSynchronize();
  cudaErrorCheck(cuda_ret, "Unable to launch kernel", 1);

  return h_fx;
}