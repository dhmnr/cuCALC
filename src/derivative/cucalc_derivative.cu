#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <stdio.h>

#include <iostream>

#include "cucalc/cucalc.h"
#include "cucalc/cucalc_common.h"
#include "cucalc/cucalc_integration.h"

__global__ void cucalc_derivative_backward(void *func, double h, double *d_fx,double *h_fx, double a, size_t thread_count) {

  size_t tid = threadIdx.x + blockIdx.x * blockDim.x;
  double x = tid*h + a; 
  double result = (cucalc_func(func))(x);
  //__syncthreads();
  d_fx[tid] = result;
  if(tid != 0){
    h_fx[tid-1] = (d_fx[tid-1] - d_fx[tid])/h;
  } 
}
__global__ void cucalc_derivative_forward(void *func, double h, double *d_fx, double a,
                                                 size_t n) {
  size_t tid = threadIdx.x + blockIdx.x * blockDim.x;

  double mult, x = tid * h + a;
  double res = (cucalc_func(func))(x);

  if (tid == 0 || tid == (n - 1))
    mult = 1 / 2;
  else
    mult = 1;
  d_fx[tid] = mult * res;
}
__global__ void cucalc_derivative_central(void *func, double h, double *d_fx, double a,
                                                 size_t n) {
  size_t tid = threadIdx.x + blockIdx.x * blockDim.x;

  double mult, x = tid * h + a;
  double res = (cucalc_func(func))(x);

  if (tid == 0 || tid == (n - 1))
    mult = 1 / 2;
  else
    mult = 1;
  d_fx[tid] = mult * res;
}

double* cucalc_derivative_backward(void *func, double a, double b, size_t steps) {
  hipError_t cuda_ret;
  int BLOCK_SIZE = 512;
  size_t thread_count = steps + 2;
  dim3 blockSize(BLOCK_SIZE, 1, 1);
  dim3 gridSize((thread_count - 1) / BLOCK_SIZE + 1, 1, 1);
  double h = (b - a) / steps;
  a = a - h; //for 1 backward

  double *d_fx, *h_fx;
  cuda_ret = hipMalloc((void **)&d_fx, sizeof(double) * thread_count);
  cudaErrorCheck(cuda_ret, "Unable to allocate memory on GPU", 1);

  cuda_ret = hipHostMalloc((void **)&h_fx, sizeof(double) * thread_count);
  cudaErrorCheck(cuda_ret, "Unable to allocate memory on host", 1);

  cucalc_derivative_backward<<<gridSize, blockSize>>>(func, h, d_fx, h_fx, a, thread_count + 1);

  cuda_ret = hipMemcpy(h_fx, d_fx, thread_count * sizeof(double), hipMemcpyDeviceToHost);
  cudaErrorCheck(cuda_ret, "Unable to memory to host", 1);

  cuda_ret = hipDeviceSynchronize();
  cudaErrorCheck(cuda_ret, "Unable to launch kernel", 1);

  return h_fx;
}

double cucalc_derivative_forward(void *func, double a, double b, size_t steps) {
  hipError_t cuda_ret;
  int BLOCK_SIZE = 512;
  size_t thread_count = steps + 2;
  dim3 blockSize(BLOCK_SIZE, 1, 1);
  dim3 gridSize((thread_count - 1) / BLOCK_SIZE + 1, 1, 1);
  double h = (b - a) / steps;

  double *d_fx, *h_fx;
  cuda_ret = hipMalloc((void **)&d_fx, sizeof(double) * thread_count);  // TODO error handling
  cudaErrorCheck(cuda_ret, "Unable to allocate memory on GPU", 1);

  cuda_ret = hipHostMalloc((void **)&h_fx, sizeof(double) * thread_count);
  cudaErrorCheck(cuda_ret, "Unable to allocate memory on host", 1);

  cucalc_derivative_forward<<<gridSize, blockSize>>>(func, h, d_fx, a, thread_count);
  cuda_ret = hipDeviceSynchronize();
  cudaErrorCheck(cuda_ret, "Unable to launch kernel", 1);

  return h * cucalc_reduction_sum(d_fx, thread_count);
}

double cucalc_derivative_central(void *func, double a, double b, size_t steps) {
  hipError_t cuda_ret;
  int BLOCK_SIZE = 512;
  size_t thread_count = steps + 2;
  dim3 blockSize(BLOCK_SIZE, 1, 1);
  dim3 gridSize((thread_count - 1) / BLOCK_SIZE + 1, 1, 1);
  double h = (b - a) / steps;

  double *d_fx, *h_fx;
  cuda_ret = hipMalloc((void **)&d_fx, sizeof(double) * thread_count);  // TODO error handling
  cudaErrorCheck(cuda_ret, "Unable to allocate memory on GPU", 1);

  cuda_ret = hipHostMalloc((void **)&h_fx, sizeof(double) * thread_count);
  cudaErrorCheck(cuda_ret, "Unable to allocate memory on host", 1);

  cucalc_derivative_backward<<<gridSize, blockSize>>>(func, h, d_fx, a, thread_count);
  cuda_ret = hipDeviceSynchronize();
  cudaErrorCheck(cuda_ret, "Unable to launch kernel", 1);

  return h * cucalc_reduction_sum(d_fx, thread_count);
}