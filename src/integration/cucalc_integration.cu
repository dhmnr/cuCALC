#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <stdio.h>

#include <iostream>

#include "cucalc/cucalc.h"
#include "cucalc/cucalc_common.h"
#include "cucalc/cucalc_integration.h"

__global__ void cucalc_integration_trapez_kernel(void *func, double h, double *d_fx, double a,
                                                 size_t n) {
  size_t tid = threadIdx.x + blockIdx.x * blockDim.x;

  double mult, x = tid * h + a;
  double res = (cucalc_func(func))(x);

  if (tid == 0 || tid == (n - 1))
    mult = 1 / 2;
  else
    mult = 1;
  d_fx[tid] = mult * res;
}

double cucalc_integration_trapez(void *func, double a, double b, size_t steps) {
  hipError_t cuda_ret;
  int BLOCK_SIZE = 512;
  size_t thread_count = steps + 2;
  dim3 blockSize(BLOCK_SIZE, 1, 1);
  dim3 gridSize((thread_count - 1) / BLOCK_SIZE + 1, 1, 1);
  double h = (b - a) / steps;

  double *d_fx, *h_fx;
  cuda_ret = hipMalloc((void **)&d_fx, sizeof(double) * thread_count);  // TODO error handling
  cudaErrorCheck(cuda_ret, "Unable to allocate memory on GPU", 1);

  cuda_ret = hipHostMalloc((void **)&h_fx, sizeof(double) * thread_count);
  cudaErrorCheck(cuda_ret, "Unable to allocate memory on host", 1);

  cucalc_integration_trapez_kernel<<<gridSize, blockSize>>>(func, h, d_fx, a, thread_count);
  cuda_ret = hipDeviceSynchronize();
  cudaErrorCheck(cuda_ret, "Unable to launch kernel", 1);

  return h * cucalc_reduction_sum(d_fx, thread_count);
}

// double cucalc_integration_qag()
