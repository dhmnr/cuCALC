#include "hip/hip_runtime.h"
#define BLOCK_SIZE 512
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <stdio.h>

#include <iostream>

#include "cucalc/cucalc.h"

// #define gpuErrchk(val) cudaErrorCheck(val, __FILE__, __LINE__, true)
void cudaErrorCheck(hipError_t err, const char *message, bool abort) {
  if (err != hipSuccess) {
    printf("%s:%s\n%s\n", hipGetErrorName(err), hipGetErrorString(err), message);
    if (abort) exit(-1);
  }
}

__global__ void reduction_sum(double *array, size_t array_length) {
  __shared__ double partialSum[2 * BLOCK_SIZE];
  unsigned int t = threadIdx.x;
  unsigned int start = 2 * blockIdx.x * blockDim.x;
  partialSum[t] = array[start + t];
  partialSum[blockDim.x + t] = array[start + blockDim.x + t];
  unsigned int offset;
  if (blockIdx.x == array_length / (BLOCK_SIZE << 1)) {
    offset = (array_length % (BLOCK_SIZE << 1));
    if (blockDim.x + t < offset) {
      partialSum[t] = array[start + t];
      partialSum[blockDim.x + t] = array[start + blockDim.x + t];
    } else if (t < offset) {
      partialSum[t] = array[start + t];
      partialSum[blockDim.x + t] = 0;
    } else {
      partialSum[t] = 0;
      partialSum[blockDim.x + t] = 0;
    }
  } else {
    partialSum[t] = array[start + t];
    partialSum[blockDim.x + t] = array[start + blockDim.x + t];
  }
  for (unsigned int stride = BLOCK_SIZE; stride >= 1; stride = stride / 2) {
    __syncthreads();
    if (t < stride) partialSum[t] += partialSum[t + stride];
  }
  if (threadIdx.x == 0) {
    array[blockIdx.x] = partialSum[0];
  }
}
/*
The following Reduction Sum function is used to calculate the sum provided the array and size of
array. The reduction kernel is launched repeatedly until it reduces to single block where the final
sum is calculated.
*/
double cucalc_reduction_sum(double *array, size_t array_length) {
  hipError_t cuda_ret;
  size_t block_count;
  double reduction_sum_output;

  for (size_t element_count = array_length; element_count > 2; element_count = block_count) {
    block_count = (element_count - 1) / (BLOCK_SIZE * 2) + 1;
    dim3 blockSize(BLOCK_SIZE, 1, 1);
    dim3 gridSize(block_count, 1, 1);

    reduction_sum<<<gridSize, blockSize>>>(array, element_count);
    cuda_ret = hipDeviceSynchronize();
    cudaErrorCheck(cuda_ret, "Unable to launch/execute reduction kernel\n", 1);
  }
  cuda_ret = hipMemcpy(&reduction_sum_output, array, sizeof(double), hipMemcpyDeviceToHost);
  cudaErrorCheck(cuda_ret, "Unable to copy reduction sum to host\n", 1);

  return reduction_sum_output;
}