#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <stdio.h>

#include "cucalc/cucalc.h"
#include "cucalc/cucalc_integration.h"

__device__ double cubed(double x) { return x * x * x; }

__device__ cucalc_func d_func = cubed;

int main(int argc, char const* argv[]) {
  void* h_func;
  hipError_t cuda_ret
      = hipMemcpyFromSymbol(&h_func, HIP_SYMBOL(d_func), sizeof(cucalc_func), 0, hipMemcpyDeviceToHost);
  if (cuda_ret != hipSuccess) {
    printf("Unable to copy device function\n");
    printf(hipGetErrorString(cuda_ret));
    printf("\n");
  }

  double result = cucalc_integration_trapez(h_func, 0, 8, 1 << 18);
  printf("Integral  = %f\n", result);

  return 0;
}
