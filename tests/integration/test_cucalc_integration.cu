#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <stdio.h>

#include "cucalc/cucalc.h"
#include "cucalc/cucalc_integration.h"

__device__ double cubed(double x) { return x * x * x; }

__device__ cucalc_func d_func = cubed;

int main(int argc, char const* argv[]) {
  hipSetDevice(3);
  void* h_func;
  hipError_t cuda_ret
      = hipMemcpyFromSymbol(&h_func, HIP_SYMBOL(d_func), sizeof(cucalc_func), 0, hipMemcpyDeviceToHost);
  if (cuda_ret != hipSuccess) {
    printf("Unable to copy device function\n");
    printf(hipGetErrorString(cuda_ret));
    printf("\n");
  }

  double result = cucalc_integration_trapez(h_func, 0, 8, 1 << 18);
  if (1024 == (int)result)
    printf("Test passed!\n");
  else
    printf("Test failed! expected : %d, actual : %d\n", 1024, (int)result);
  return 0;
}
