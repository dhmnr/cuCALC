#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <stdio.h>

#include "cucalc/cucalc.h"
#include "cucalc/cucalc_derivative.h"

__device__ double squared(double x) { return x * x; }

__device__ cucalc_func d_func = squared;

int main(int argc, char const* argv[]) {
  hipSetDevice(3);
  void* h_func;
  hipError_t cuda_ret
      = hipMemcpyFromSymbol(&h_func, HIP_SYMBOL(d_func), sizeof(cucalc_func), 0, hipMemcpyDeviceToHost);
  if (cuda_ret != hipSuccess) {
    printf("Unable to copy device function\n");
    printf(hipGetErrorString(cuda_ret));
    printf("\n");
  }

  double *result_backward = cucalc_derivative_backward(h_func, 0, 2, 1000);
  printf("Backward %f\n",result_backward[500]);
  
  double *result_forward = cucalc_derivative_forward(h_func, 0, 2, 1000);
  printf("Forward %f\n",result_forward[500]);
  
  double *result_central = cucalc_derivative_central(h_func, 0, 2, 1000);
  printf("Central %f\n",result_central[500]);
  // if (1024 == (int)result)
  //   printf("Test passed!\n");
  // else
  //   printf("Test failed! expected : %d, actual : %d\n", 1024, (int)result);
  // return 0;
}